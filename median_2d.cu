#include "hip/hip_runtime.h"
/**
* @file    2D Median filterusing NVIDIA CUDA
* @author  Institute for Photon Science and Synchrotron Radiation, Karlsruhe Institute of Technology
*
* @date    2015-2018
* @version 0.5.0
*
*
* @section LICENSE
*
* This program is copyrighted by the author and Institute for Photon Science and Synchrotron Radiation,
* Karlsruhe Institute of Technology, Karlsruhe, Germany;
*
* The current implemetation contains the following licenses:
*
* 1. TinyXml package:
*      Original code (2.0 and earlier )copyright (c) 2000-2006 Lee Thomason (www.grinninglizard.com). <www.sourceforge.net/projects/tinyxml>.
*      See src/utils/tinyxml.h for details.
*
*/

#include <>

#define __HIPCC__

#include <hip/device_functions.h>
#include <math_functions.h>

#include "src/data_types/data_structs.h"

//#define IND(X, Y, Z) (((Z) * container_size.height + (Y)) * (container_size.pitch / sizeof(float)) + (X)) 
//#define SIND(X, Y, Z) ((((Z) + radius_2) * shared_block_size.y + ((Y) + radius_2)) * shared_block_size.x + ((X) + radius_2))

#define IND(X, Y) ((Y) * (container_size.pitch / sizeof(float)) + (X)) 
#define SIND(X, Y) ((((Y) + radius_2)) * shared_block_size.x + ((X) + radius_2))

__constant__ DataSize3 container_size;


extern __shared__ float shared[];

__device__ void sort(float* buffer, size_t length)
{
  for (int i = 0; i < length - 1; i++) {
    for (int k = 0; k < length - i - 1; k++) {
      if (buffer[k] > buffer[k + 1]) {
        float a = buffer[k];
        buffer[k] = buffer[k + 1];
        buffer[k + 1] = a;
      }
    }
  }
}

/* See a note about the thread block size in cuda_operation_median.cpp file.*/
extern "C" __global__ void median_2d(
  const float* input,
        size_t width,
        size_t height,
        size_t radius,
        float* output)
{
  int radius_2 = radius / 2;

  dim3 shared_block_size(
    blockDim.x + 2 * radius_2,
    blockDim.y + 2 * radius_2
   );

  dim3 global_id(
    blockDim.x * blockIdx.x + threadIdx.x,
    blockDim.y * blockIdx.y + threadIdx.y
   );

  /* Load data to the shared memoty */
  size_t global_x = global_id.x < width ? global_id.x : 2 * width - global_id.x - 2;
  size_t global_y = global_id.y < height ? global_id.y : 2 * height - global_id.y - 2;

  /* Main area */
  shared[SIND(threadIdx.x, threadIdx.y)] = input[IND(global_x, global_y)];

  /* Left slice */
  if (threadIdx.x < radius_2) {
    int offset = blockDim.x * blockIdx.x - radius_2 + threadIdx.x;
    size_t global_x_l = offset >= 0 ? offset : -offset;
    shared[SIND(-radius_2 + threadIdx.x, threadIdx.y)] = input[IND(global_x_l, global_y)];
  }

  /* Right slice */
  if (threadIdx.x > blockDim.x - 1 - radius_2) {
    int index = blockDim.x - threadIdx.x;
    int offset = blockDim.x *(blockIdx.x + 1) + radius_2 - index;
    size_t global_x_r = offset < width ? offset : 2 * width - offset - 2;
    shared[SIND(radius_2 + threadIdx.x, threadIdx.y)] = input[IND(global_x_r, global_y)];
  }

  /* Upper slice */
  if (threadIdx.y < radius_2) {
    int offset = blockDim.y * blockIdx.y - radius_2 + threadIdx.y;
    size_t global_y_u = offset >= 0 ? offset : -offset;
    shared[SIND(threadIdx.x, -radius_2 + threadIdx.y)] = input[IND(global_x, global_y_u)];
  }

  /* Bottom slice */
  if (threadIdx.y > blockDim.y - 1 - radius_2) {
    int index = blockDim.y - threadIdx.y;
    int offset = blockDim.y *(blockIdx.y + 1) + radius_2 - index;
    size_t global_y_b = offset < height ? offset : 2 * height - offset - 2;
    shared[SIND(threadIdx.x, radius_2 + threadIdx.y)] = input[IND(global_x, global_y_b)];
  }


  
  // <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<< //
  // -------------------------------------------------------- //
  __syncthreads();
  // -------------------------------------------------------- //
  // <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<< //

  if (global_id.x < width && global_id.y < height) {
    float buffer[49]; /* Max supported radius is 7, we have to store 7*7*7 values. */
      for (size_t iy = 0; iy < radius; ++iy) {
        for (size_t ix = 0; ix < radius; ++ix) {
          size_t lx = threadIdx.x - ix + radius_2;
          size_t ly = threadIdx.y - iy + radius_2;
          buffer[iy * radius + ix] = shared[SIND(lx, ly)];
        }
      }
    

    size_t length = radius * radius;
    sort(buffer, length);

    output[IND(global_id.x, global_id.y)] = buffer[length / 2];
  }
}